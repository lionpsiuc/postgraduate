#include "hip/hip_runtime.h"
//=============================================================================================
// Name        		: thrustRandomVectorsComparison.cu
// Author      		: Jose Refojo
// Version     		:	05-03-2012
// Creation date	:	05-03-2012
// Copyright		: Copyright belongs to Trinity Centre for High Performance Computing
// Description		: This program will compare different random number generators
//=============================================================================================

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <cstdlib>

#include <algorithm>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>

// This is the include for the host hiprand
# include <hiprand.h>

#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

using namespace std;
using std::cout;
using std::endl;

int main(void) {
	int N = 32 << 20;

	hipEvent_t thrustStart, thrustFinish;
	hipEvent_t cuRandStart, cuRandFinish;
	hipEvent_t stlStart, stlFinish;
	float thrustElapsedTime;
	float cuRandElapsedTime;
	float stlElapsedTime;

	hipEventCreate(&thrustStart);
	hipEventCreate(&thrustFinish);
	hipEventCreate(&cuRandStart);
	hipEventCreate(&cuRandFinish);
	hipEventCreate(&stlStart);
	hipEventCreate(&stlFinish);

	//////////////////////////////////////////////////////////////////////////////////////////////
	// Step 1: Generate 32M random numbers on the host with thrust
	hipEventRecord(thrustStart, 0); // We use 0 here because it is the "default" stream
	thrust::host_vector<int> thrustVector(N);
	thrust::generate(thrustVector.begin(), thrustVector.end(), rand);
	hipEventRecord(thrustFinish, 0);

	//////////////////////////////////////////////////////////////////////////////////////////////
	// Step 2: Generate 32M random numbers on the host with cuRand
	// pointers to host memory
	float *host_cuRandVector;
	// pointers to device memory
	float *device_cuRandVector;

	hipEventRecord(cuRandStart, 0); // We use 0 here because it is the "default" stream

	host_cuRandVector = (float*) malloc(N*sizeof(float));
	hipMalloc ((void **) &device_cuRandVector, sizeof(float)*N);
	// Declare and initialize the pseudo-random number generator
	hiprandGenerator_t generator;
	hiprandCreateGenerator (&generator,HIPRAND_RNG_PSEUDO_DEFAULT);
	// Set the seed
	hiprandSetPseudoRandomGeneratorSeed (generator,1234ULL);
	// Generate N random numbers from an uniform distribution
	hiprandGenerateUniform (generator, device_cuRandVector , N);
	// Copy data from device memory to host memory
	hipMemcpy(host_cuRandVector, device_cuRandVector, sizeof(float)*N, hipMemcpyDeviceToHost);

	hipEventRecord(cuRandFinish, 0); // We use 0 here because it is the "default" stream
	//////////////////////////////////////////////////////////////////////////////////////////////
	// Step 3: same thing with stl
	hipEventRecord(stlStart, 0); // We use 0 here because it is the "default" stream
	std::vector<float> stlVector;
	for (int i=0;i<N;i++) {
		stlVector.push_back(rand() / (float)RAND_MAX);
	}
	hipEventRecord(stlFinish, 0);

	// Print c
	unsigned int ui;
	unsigned int printIterations = min(20,(int)(thrustVector.size()));	

	cout << "thrustVector = ";
	for (ui=0;ui<printIterations;ui++) {
		cout << thrustVector[ui] << " ";
	}
	cout << endl;

	cout << "cuRandVector = ";
	for (ui=0;ui<printIterations;ui++) {
		cout << host_cuRandVector[ui] << " ";
	}
	cout << endl;

	cout << "stlVector = ";
	for (ui=0;ui<printIterations;ui++) {
		cout << stlVector[ui] << " ";
	}
	cout << endl;

	hipEventSynchronize(thrustStart);  // This is optional, we shouldn't need it
	hipEventSynchronize(thrustFinish); // This isn't - we need to wait for the event to finish
	hipEventElapsedTime(&thrustElapsedTime, thrustStart, thrustFinish);
	hipEventSynchronize(cuRandStart);  // This is optional, we shouldn't need it
	hipEventSynchronize(cuRandFinish); // This isn't - we need to wait for the event to finish
	hipEventElapsedTime(&cuRandElapsedTime, cuRandStart, cuRandFinish);
	hipEventSynchronize(stlStart);  // This is optional, we shouldn't need it
	hipEventSynchronize(stlFinish); // This isn't - we need to wait for the event to finish
	hipEventElapsedTime(&stlElapsedTime, stlStart, stlFinish);

	cout << "Total time with thrust: " << thrustElapsedTime << " with cuRand: "<< cuRandElapsedTime << " on the host: " << stlElapsedTime << endl; 
	free(host_cuRandVector);
	hipFree (device_cuRandVector);

	return 0;
}
