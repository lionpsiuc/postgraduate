//=============================================================================================
// Name        		: sharedMemoryDynamic.cu
// Author      		: Jose Refojo
// Version     		:	20-03-2014
// Creation date	:	06-02-2013
// Copyright		: Copyright belongs to Trinity Centre for High Performance Computing
// Description		: This program shows how to use the shared memory
//=============================================================================================

#define BLOCK_SIZE 8

#include <hip/hip_runtime.h>
#include "stdio.h"


// Shared memory is shared by all the threads in a block, and it is initialized (and used) inside a global
// or device function - we will use a global function here but the usage in the device functions is exactly
// the same
__global__ void scanTheadInformationGPU(int *threadIdsGPU,int *threadOtherIdsGPU,int *blockIdsGPU,int Ntot) {
	int idx=blockIdx.x*blockDim.x+threadIdx.x;

	extern __shared__ int sharedMemoryThread[];
        //__shared__ float sharedMemoryThread[BLOCK_SIZE];
	sharedMemoryThread[threadIdx.x] = threadIdx.x;

        __syncthreads();

	if ( idx <Ntot ) {
		threadIdsGPU[idx]=sharedMemoryThread[threadIdx.x];
		threadOtherIdsGPU[idx]=sharedMemoryThread[threadIdx.x]+blockIdx.x*blockDim.x;
		blockIdsGPU[idx]=blockIdx.x;
	}
}


int main() {
	// pointers to host memory
	int *threadIds,*threadOtherIds, *blockIds;
	// pointers to device memory
	int *threadIdsGPU,*threadOtherIdsGPU, *blockIdsGPU;
	// N is the total size that we want
	int N=18;
	int i;

	// Allocate arrays threadIds and blockIds on host
	threadIds = (int*) malloc(N*sizeof(int));
	threadOtherIds = (int*) malloc(N*sizeof(int));
	blockIds = (int*) malloc(N*sizeof(int));

	// Allocate arrays threadIdsGPU and blockIdsGPU on device
	hipMalloc ((void **) &threadIdsGPU, sizeof(int)*N);
	hipMalloc ((void **) &threadOtherIdsGPU, sizeof(int)*N);
	hipMalloc ((void **) &blockIdsGPU, sizeof(int)*N);

/*
	// Copy data from host memory to device memory (not needed, but this is how you do it)
	cudaMemcpy(threadIdsGPU, threadIds, sizeof(int)*N, cudaMemcpyHostToDevice);
	cudaMemcpy(blockIdsGPU, blockIds, sizeof(int)*N, cudaMemcpyHostToDevice);
*/

	// Compute the execution configuration
	int block_size=BLOCK_SIZE;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	// Scan information from the threads
	scanTheadInformationGPU<<<dimGrid,dimBlock,BLOCK_SIZE>>>(threadIdsGPU,threadOtherIdsGPU, blockIdsGPU, N);

	// Copy data from device memory to host memory
	hipMemcpy(threadIds, threadIdsGPU, sizeof(int)*N, hipMemcpyDeviceToHost);
	hipMemcpy(threadOtherIds, threadOtherIdsGPU, sizeof(int)*N, hipMemcpyDeviceToHost);
	hipMemcpy(blockIds, blockIdsGPU, sizeof(int)*N, hipMemcpyDeviceToHost);

	// Print all the data about the threads
	printf(" dimGrid=%d\n",dimGrid.x);
	for (i=0; i<N; i++) {
	       printf(" threadIds[%d]=%d\n",i,threadIds[i]);
	}
	for (i=0; i<N; i++) {
	       printf(" threadOtherIds[%d]=%d\n",i,threadOtherIds[i]);
	}
	for (i=0; i<N; i++) {
	       printf(" blockIds[%d]=%d\n",i,blockIds[i]);
	}

	// Free the memory
	free(threadIds);
	free(threadOtherIds);
	free(blockIds); 

	hipFree(threadIdsGPU);
	hipFree(threadOtherIdsGPU);
	hipFree(blockIdsGPU);
}
