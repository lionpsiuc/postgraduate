//=============================================================================================
// Name        		: cudaStreamsCallback.cu
// Author      		: Jose Refojo
// Version     		:	16-02-16
// Creation date	:	16-02-16
// Copyright		: Copyright belongs to Trinity Centre for High Performance Computing
// Description		: This program shows how to use callbacks with cuda Streams
//=============================================================================================


#include <hip/hip_runtime.h>
#include "stdio.h"


void CUDART_CB MyCustomCallback(hipStream_t stream, hipError_t status, void *data){
    printf("Inside callback: %s\n", (char *)data);
}

__global__ void scanThreadInformationGPU(int *threadIdsGPU, int *blockIdsGPU,int Ntot) {
	int idx=blockIdx.x*blockDim.x+threadIdx.x;

#ifdef WITH_MY_DEBUG
	printf ("cudaStreams::scanThreadInformationGPU blockIdx.x=%d  threadIdx.x=%d\n",blockIdx.x,threadIdx.x);
#endif
	if ( idx <Ntot ) {
		threadIdsGPU[idx]=threadIdx.x;
	}
}

__global__ void scanBlockInformationGPU(int *threadIdsGPU, int *blockIdsGPU,int Ntot) {
	int idx=blockIdx.x*blockDim.x+threadIdx.x;

#ifdef WITH_MY_DEBUG
	printf ("cudaStreams::scanBlockInformationGPU blockIdx.x=%d  threadIdx.x=%d\n",blockIdx.x,threadIdx.x);
#endif
	if ( idx <Ntot ) {
		blockIdsGPU[idx]=blockIdx.x;
	}
}

bool verbose=false;

int main() {
	int i;

	// pointers to host memory
	int *threadIds, *blockIds;
	// pointers to device memory
	int *threadIdsGPU, *blockIdsGPU;
	// N is the total size that we want
	int N=10;

	// Declare and allocate two streams
	hipStream_t stream[2]; 
	for (i = 0; i < 2; ++i) {
		hipStreamCreate(&stream[i]);
	}

	// Allocate arrays threadIds and blockIds on host
	threadIds = (int*) malloc(N*sizeof(int));
	blockIds = (int*) malloc(N*sizeof(int));

	// Allocate arrays threadIdsGPU and blockIdsGPU on device
	hipMalloc ((void **) &threadIdsGPU, sizeof(int)*N);
	hipMalloc ((void **) &blockIdsGPU, sizeof(int)*N);
/*
	// Copy data from host memory to device memory (not needed, but this is how you do it)
	cudaMemcpy(threadIdsGPU, threadIds, sizeof(int)*N, cudaMemcpyHostToDevice);
	cudaMemcpy(blockIdsGPU, blockIds, sizeof(int)*N, cudaMemcpyHostToDevice);
*/

	// Copy data from host memory to device memory (not needed, but this is how you do it)
	// Stream 0
	hipMemcpyAsync(threadIdsGPU, threadIds, sizeof(int)*N, hipMemcpyHostToDevice,stream[0]);
	hipStreamAddCallback(stream[0], MyCustomCallback, (void*)("Stream 0: hipMemcpyAsync - hipMemcpyHostToDevice has finished\n"),0);

	// Stream 1
	hipMemcpyAsync(blockIdsGPU, blockIds, sizeof(int)*N, hipMemcpyHostToDevice,stream[1]);


	// Compute the execution configuration
	int block_size=5;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	// Scan information from the threads
	scanThreadInformationGPU<<<dimGrid,dimBlock,0,stream[0]>>>(threadIdsGPU, blockIdsGPU, N);
	hipStreamAddCallback(stream[0], MyCustomCallback, (void*)("Stream 0: scanThreadInformationGPU has finished\n"),0);
	scanBlockInformationGPU<<<dimGrid,dimBlock,0,stream[1]>>>(threadIdsGPU, blockIdsGPU, N);

	// Copy data from device memory to host memory
	hipMemcpyAsync(threadIds, threadIdsGPU, sizeof(int)*N, hipMemcpyDeviceToHost,stream[0]);
	hipStreamAddCallback(stream[0], MyCustomCallback, (void*)("Stream 0: hipMemcpyAsync - hipMemcpyDeviceToHost has finished\n"),0);
	hipMemcpyAsync(blockIds, blockIdsGPU, sizeof(int)*N, hipMemcpyDeviceToHost,stream[1]);

	// Print all the data about the threads

	if (verbose) {
		printf(" dimGrid=%d\n",dimGrid.x);
		for (i=0; i<N; i++) {
		       printf(" threadIds[%d]=%d\n",i,threadIds[i]);
		}
		for (i=0; i<N; i++) {
		       printf(" blockIds[%d]=%d\n",i,blockIds[i]);
		}
	}

	for (int i = 0; i < 2; ++i) 
		hipStreamDestroy(stream[i]); 

	// Free the memory
	free(threadIds);
	free(blockIds); 

	hipFree(threadIdsGPU);
	hipFree(blockIdsGPU);
}
