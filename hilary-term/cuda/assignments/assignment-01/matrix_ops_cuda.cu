#include "hip/hip_runtime.h"
#include "matrix_ops_cuda.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA error checking macro
#define cudaCheckError()                                                       \
  {                                                                            \
    hipError_t err = hipGetLastError();                                      \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__,        \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

// Setup the CUDA device
extern "C" int setupCuda() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    fprintf(stderr, "Error: No CUDA devices found\n");
    return -1;
  }

  int device = 0; // Use the first device by default
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  printf("Using CUDA device: %s\n", deviceProp.name);

  hipSetDevice(device);
  cudaCheckError();

  return device;
}

// Kernel for computing row sums
__global__ void computeRowSumsKernel(float *matrix_flat, float *rowSums, int n,
                                     int m) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n) {
    float sum = 0.0f;
    for (int j = 0; j < m; j++) {
      sum += fabsf(matrix_flat[row * m + j]);
    }
    rowSums[row] = sum;
  }
}

// Kernel for computing column sums
__global__ void computeColumnSumsKernel(float *matrix_flat, float *colSums,
                                        int n, int m) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (col < m) {
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
      sum += fabsf(matrix_flat[i * m + col]);
    }
    colSums[col] = sum;
  }
}

// Kernel for reducing a vector
__global__ void reduceKernel(float *vector, float *result, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    atomicAdd(result, vector[tid]);
  }
}

// Function to flatten a 2D array for CUDA
float *flattenMatrix(float **matrix, int n, int m) {
  float *flat = (float *)malloc(n * m * sizeof(float));
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < m; j++) {
      flat[i * m + j] = matrix[i][j];
    }
  }
  return flat;
}

// Compute row sums on GPU
extern "C" float *computeRowSumsGPU(float **matrix, int n, int m,
                                    int threads_per_block) {
  // Flatten the matrix for GPU
  float *matrix_flat = flattenMatrix(matrix, n, m);

  // Allocate device memory
  float *d_matrix, *d_rowSums;
  hipMalloc((void **)&d_matrix, n * m * sizeof(float));
  cudaCheckError();
  hipMalloc((void **)&d_rowSums, n * sizeof(float));
  cudaCheckError();

  // Copy matrix to device
  hipMemcpy(d_matrix, matrix_flat, n * m * sizeof(float),
             hipMemcpyHostToDevice);
  cudaCheckError();

  // Setup grid and blocks
  int numBlocks = (n + threads_per_block - 1) / threads_per_block;

  // Launch kernel
  computeRowSumsKernel<<<numBlocks, threads_per_block>>>(d_matrix, d_rowSums, n,
                                                         m);
  cudaCheckError();

  // Allocate host memory for result
  float *rowSums = (float *)malloc(n * sizeof(float));

  // Copy result back to host
  hipMemcpy(rowSums, d_rowSums, n * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError();

  // Free device memory
  hipFree(d_matrix);
  hipFree(d_rowSums);
  free(matrix_flat);

  return rowSums;
}

// Compute column sums on GPU
extern "C" float *computeColumnSumsGPU(float **matrix, int n, int m,
                                       int threads_per_block) {
  // Flatten the matrix for GPU
  float *matrix_flat = flattenMatrix(matrix, n, m);

  // Allocate device memory
  float *d_matrix, *d_colSums;
  hipMalloc((void **)&d_matrix, n * m * sizeof(float));
  cudaCheckError();
  hipMalloc((void **)&d_colSums, m * sizeof(float));
  cudaCheckError();

  // Copy matrix to device
  hipMemcpy(d_matrix, matrix_flat, n * m * sizeof(float),
             hipMemcpyHostToDevice);
  cudaCheckError();

  // Setup grid and blocks
  int numBlocks = (m + threads_per_block - 1) / threads_per_block;

  // Launch kernel
  computeColumnSumsKernel<<<numBlocks, threads_per_block>>>(d_matrix, d_colSums,
                                                            n, m);
  cudaCheckError();

  // Allocate host memory for result
  float *colSums = (float *)malloc(m * sizeof(float));

  // Copy result back to host
  hipMemcpy(colSums, d_colSums, m * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError();

  // Free device memory
  hipFree(d_matrix);
  hipFree(d_colSums);
  free(matrix_flat);

  return colSums;
}

// Reduce vector on GPU
extern "C" float reduceGPU(float *vector, int size, int threads_per_block) {
  // Allocate device memory
  float *d_vector, *d_result;
  hipMalloc((void **)&d_vector, size * sizeof(float));
  cudaCheckError();
  hipMalloc((void **)&d_result, sizeof(float));
  cudaCheckError();

  // Initialize result to 0
  float zero = 0.0f;
  hipMemcpy(d_result, &zero, sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError();

  // Copy vector to device
  hipMemcpy(d_vector, vector, size * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError();

  // Setup grid and blocks
  int numBlocks = (size + threads_per_block - 1) / threads_per_block;

  // Launch kernel
  reduceKernel<<<numBlocks, threads_per_block>>>(d_vector, d_result, size);
  cudaCheckError();

  // Copy result back to host
  float result;
  hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError();

  // Free device memory
  hipFree(d_vector);
  hipFree(d_result);

  return result;
}
